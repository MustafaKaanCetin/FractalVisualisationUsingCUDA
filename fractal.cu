#include <hip/hip_runtime.h>

__device__ int map(int value, int start1, int stop1, int start2, int stop2) {
    return start2 + (stop2 - start2) * ((value - start1) / (float)(stop1 - start1));
}

__global__ void fractal_kernel(int width, int height, int max_iter, unsigned char* image, float zoom, float centerX, float centerY) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && y < height) {
        float zx, zy, cX, cY;
        zx = zy = 0.0f;
        cX = (x - width / 2.0f) * 4.0f / (width * zoom) + centerX;
        cY = (y - height / 2.0f) * 4.0f / (height * zoom) + centerY;
        float zx2 = 0.0f, zy2 = 0.0f;
        int iter = 0;
        while (zx * zx + zy * zy < 4.0f && iter < max_iter) {
            zy = 2.0f * zx * zy +cY;
            zx = zx2 - zy2 + cX;
            zx2 = zx * zx;
            zy2 = zy * zy;
            iter++;
        }

        int idx = (y * width + x) * 3;
        int bright = map(iter, 0, max_iter, 0, 255);

        if((iter >= max_iter - 5) || (bright < 10)) {
            bright = 0;
        }

        int red = map(bright * bright, 0, 6502, 0, 255);
        int green = bright;
        int blue = map(sqrtf(bright), 0, sqrtf(255), 0, 255);

        image[idx] = red;
        image[idx + 1] = green;
        image[idx + 2] = blue;
    }

}

extern "C" void launch_kernel(int width, int height, int max_iter, unsigned char* image, float zoom, float centerX, float centerY) {
    unsigned char* d_image;
    size_t image_size = width * height * 3 * sizeof(unsigned char);
    hipMalloc(&d_image, image_size);
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    fractal_kernel<<<numBlocks, threadsPerBlock>>>(width, height, max_iter, d_image, zoom, centerX, centerY);

    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
    hipFree(d_image);
}